#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////////////////
// Converted GS Algorithm for MRI Image Reconstruction to CUDA //
//															   //
// Authors: NGUYEN Hong Quan								   //
//			TRAN Nguyen Phuong Trinh						   //
//															   //
// Emails: nguyenhongquan_eeit13@hotmail.com				   //
//		   trinhtran2151995@gmail.com						   //
//															   //
// Date: 8th October, 2016									   //
/////////////////////////////////////////////////////////////////
#include "hip/hip_runtime.h"
#include ""
#include <arrayfire.h>
#include <af/hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <thread>
#include <stdio.h>
#include <math.h>
#include <chrono>
#include <hipfft/hipfft.h>
#include <string>
#include <hipblas.h>
////////////////////////////////////////////////
using namespace std;
////////////////////////////////////////////////
const char kMaskFilePath[] = "E:\\MRI\\workspace\\Datas\\kmask1.txt";
const char dataFilePath[] = "E:\\MRI\\workspace\\Datas\\underSampledData1.txt";
const char IrefPath[] = "E:\\MRI\\workspace\\Datas\\irefData.txt";
string HroGSPath = "E:\\MRI\\workspace\\Results\\HroGS";
////////////////////////////////////////////////
const int IterNumCG = 250;
int NFrame, NSlice, NY, NX, NSamples;
float **cpu_data, **cpu_rhogs;
float *cpu_Iref;
int *cpu_kMaskIndex;
////////////////////////////////////////////////
af::array Apk, alpha, rsnew, p, r, c_gs, rsold, gpu_Iref, gpu_kMaskIndex, gpu_FrameDatas, idx, gpu_rhogs;
////////////////////////////////////////////////
void readData();
void writeResult_RhoGS();
////////////////////////////////////////////////
int main()
{
	af::setDevice(0);
	af_info();
	auto FrameCGFunction = [](){
		auto fftshift3D = [](af::array in){
			return af::shift(in, (int)in.dims(0) / 2, (int)in.dims(1) / 2, (int)in.dims(2) / 2);
		};

		auto ifftshift3D = [](af::array in){
			return af::shift(in, ((int)in.dims(0) + 1) / 2, ((int)in.dims(1) + 1) / 2, ((int)in.dims(2) + 1) / 2);
		};

		for (int id = 0; id < NFrame; id++)
		{
			r = gpu_FrameDatas(af::span, af::span, af::span, id);
			r = af::flat(r);
			p = r;
			rsold = af::dot(r, r, AF_MAT_CONJ, AF_MAT_NONE);
			c_gs = af::constant(0, r.dims(0), c32);
			for (int i = 0; i < IterNumCG; i++)
			{
				Apk = af::moddims(p, NSlice, NY, NX, 1);
				Apk = ifftshift3D(af::fft3(fftshift3D(gpu_Iref*ifftshift3D(af::ifft3(fftshift3D(Apk)))))); Apk.eval();
				Apk = af::flat(Apk)*idx;
				alpha = af::real((rsold) / (af::dot(p, Apk, AF_MAT_CONJ, AF_MAT_NONE))); alpha.eval();
				c_gs += af::tile(alpha, p.dims(0))*p; c_gs.eval();
				r -= af::tile(alpha, Apk.dims(0))*Apk; r.eval();
				rsnew = af::dot(r, r, AF_MAT_CONJ, AF_MAT_NONE);
				p = r + af::tile(rsnew / rsold, p.dims(0))*p; p.eval();
				rsold = rsnew;
				af::sync();
			}
			c_gs *= idx;
			c_gs = af::moddims(c_gs, NSlice, NY, NX, 1);
			gpu_rhogs = af::abs(gpu_Iref*ifftshift3D(af::ifft3(fftshift3D(c_gs))));
			gpu_rhogs.host(cpu_rhogs[id]);
			af::sync();
		}
	};
	auto start = chrono::high_resolution_clock::now();
	readData();
	auto stop = chrono::high_resolution_clock::now();
	cout << "readData() take " << chrono::duration_cast<chrono::milliseconds>(stop - start).count() << ".ms" << endl;
	start = chrono::high_resolution_clock::now();
	FrameCGFunction();
	stop = chrono::high_resolution_clock::now();
	cout << "process() take " << chrono::duration_cast<chrono::milliseconds>(stop - start).count() << ".ms" << endl;
	start = chrono::high_resolution_clock::now();
	writeResult_RhoGS();
	stop = chrono::high_resolution_clock::now();
	cout << "writeResult() take " << chrono::duration_cast<chrono::milliseconds>(stop - start).count() << ".ms" << endl;
	hipDeviceSynchronize();
	hipDeviceReset();
	free(cpu_kMaskIndex); free(cpu_Iref);
	for (int i = 0; i < NFrame; i++)
	{
		free(cpu_data[i]);
		free(cpu_rhogs[i]);
	}
	free(cpu_data);	free(cpu_rhogs);
	return 0;
}
////////////////////////////////////////////////
void readData()
{
	FILE *dataFile, *kMaskFile, *IrefFile;
	dataFile = fopen(dataFilePath, "r");
	kMaskFile = fopen(kMaskFilePath, "r");
	IrefFile = fopen(IrefPath, "r");

	//read kMask
	fscanf(kMaskFile, "%d", &NSamples);
	cpu_kMaskIndex = (int*)calloc(NSamples, sizeof(int));
	for (int i = 0; i < NSamples; i++)
	{
		fscanf(kMaskFile, "%d\n", &cpu_kMaskIndex[i]);
		cpu_kMaskIndex[i] -= 1;
	}
	gpu_kMaskIndex = af::array(NSamples, cpu_kMaskIndex, afHost);

	//read data
	fscanf(dataFile, "%d %d %d %d\n", &NFrame, &NSlice, &NY, &NX);
	cpu_data = (float**)malloc(sizeof(float*)*NFrame);
	cpu_rhogs = (float**)malloc(sizeof(float*)*NFrame);
	for (int i = 0; i < NFrame; i++)
	{
		cpu_data[i] = (float*)calloc(NSlice*NY*NX * 2, sizeof(float));
		cpu_rhogs[i]= (float*)calloc(NSlice*NY*NX, sizeof(float));
	}
	for (int i = 0; i < NFrame; i++)
	{
		for (int j = 0; j < NSamples; j++)
		{
			fscanf(dataFile, "%f %f\n", &(cpu_data[i][cpu_kMaskIndex[j] * 2]),
				&(cpu_data[i][cpu_kMaskIndex[j] * 2 + 1]));
		}
	}

	// copy host data -> GPU
	gpu_FrameDatas = af::array(NSlice, NY, NX, NFrame, c32);
	for (int i = 0; i < NFrame; i++)
	{
		gpu_FrameDatas(af::span, af::span, af::span, i) = af::array(NSlice, NY, NX, (af::cfloat*) cpu_data[i]);
	}

	//read Iref
	cpu_Iref = (float*)calloc(NSlice*NY*NX, sizeof(float));
	for (int i = 0; i < NSlice*NX*NY; i++)
	{
		fscanf(IrefFile, "%f", &cpu_Iref[i]);
	}
	gpu_Iref = af::array(NSlice, NY, NX, cpu_Iref, afHost);

	idx = af::constant(0, NSlice, NY, NX, c32);
	idx = af::flat(idx);
	idx(gpu_kMaskIndex) = 1;

	// close data files
	fclose(dataFile); fclose(kMaskFile); fclose(IrefFile);
}
//////////////////////////////////////////////////
void writeResult_RhoGS()
{
	for (int i = 0; i < NFrame; i++)
	{
		FILE* resultFile = fopen((HroGSPath + to_string(i) + string(".txt")).c_str(), "w");
		for (int j = 0; j < NSlice*NY*NX; j++)
		{
			fprintf(resultFile, "%f\n", cpu_rhogs[i][j]);
		}
		fclose(resultFile);
	}
}
/////////////////////////////////////////////////



